#include "hip/hip_runtime.h"
#include "color_d.h"

#include <cstdlib>
#include <cstring>
#include <cmath>
#include <fstream>
#include "ext_math.h"

using namespace ext_math;
using namespace std;

namespace vision {

const int FieldColorDetector::pixelSpacing=16;	//nur jeden 16ten Pixel in x- und y-Richtung scannen
const int FieldColorDetector::minFieldArea=150; //min amount of pixels for initial green seed / carpet color detection
const int FieldColorDetector::colorBorder=8;	//exclude extrem color values (0+colorBorder) < color < (255-colorBorder)
const float FieldColorDetector::greenGain=2.0;	//gain for green threshold in maybeGreen function
const float FieldColorDetector::thetas[62]={-0.003928473492851304, 0.3267591297421786, 0.15024038619767252, -0.0026580701516830778, -0.06626938819648565,
											-0.0918234800891045, -0.007024391380169659, -0.19444191475988953, 0.10805971362794498, -0.013870596388515415,
											0.020367431009978596, 0.5006415354683456, 0.0523626396220556, 0.06796659398791673, 0.13091006086089688,
											-0.04015537555250251, 0.2534669682609816, 0.22259232118528924, 0.08825125863663277, -0.06163688650966539,
											0.18292785975365364, 0.18490873559957913, 0.1308039353774413, 0.15043700747876884, -0.030408070393040373,
											-0.18234162918227365, 0.30516577669883815, -0.060749446765493896, 0.473586856960429, 0.31872308251277015,
											-0.04073046667475529, -0.157079077089501, -0.4325757850385499, 0.04335112440158662, 0.05425490604442203,
											-0.21977115232887673, 0.06711078780969373, -0.08743680491075988, -0.07236939332440999, 0.03686565322899083,
											-0.0034490299130395608, -0.1224550945463936, 0.14004650713616937, 0.053306649062165465, 0.20536885019856824,
											0.010868449852810068, 0.14466029588975235, 0.08223414822611032, 0.09088654555978222, 0.12192354321809107,
											-0.15900537301205442, -0.07653902693746543, -0.13043295430645066, 0.10422567492674986, 0.12357034980655468,
											0.06425325379992515, 0.1547906519403598, -0.11945473558477798, -0.15195968390312445, -0.060779372474056875,
											-0.06570198431532723, 0.07563040487570107};

__global__ void makeHistCr(const uint8_t * const img, int w, int h, int width, int height, int *histCr, int seedSearchBorder, int pixelSpacing){
    int idx = blockIdx.x*blockDim.x*blockDim.y + (threadIdx.x + threadIdx.y * blockDim.x);
	
	if( idx >= w*h){
		return;
	}
	int x = seedSearchBorder + (idx - idx/w*w)  * pixelSpacing;
    int y = seedSearchBorder + (idx/w ) * pixelSpacing;
	int cr = img[((x + y * width) << 1) | 3];
	// atomic operation
	atomicAdd(&histCr[cr], 1);
}

__global__ void makeHistCb(const uint8_t * const img, int w, int h, int width, int height, int *histCb, int seedCr, int pixelSpacing){
    int idx = blockIdx.x*blockDim.x*blockDim.y + (threadIdx.x + threadIdx.y * blockDim.x);
	if( idx >= w*h){
		return;
	}
	int x = (idx - idx/w*w)  * pixelSpacing;
    int y = idx/w * pixelSpacing;
	int cr = img[((x + y * width) << 1) | 3];
    if(abs(cr-seedCr)<4){
        int cb=img[(((x + y * width) >> 1) << 2) + 1];
		atomicAdd(&histCb[cb], 1);
    }
}

__global__ void makeHistY(const uint8_t * const img, int w, int h, int width, int height, int *histY, int seedCr, int seedCb, int pixelSpacing){
    int idx = blockIdx.x*blockDim.x*blockDim.y + (threadIdx.x + threadIdx.y * blockDim.x);
	if( idx >= w*h){
		return;
	}
	int x = (idx - idx/w*w)  * pixelSpacing;
    int y = idx/w * pixelSpacing;
	int cr = img[((x + y * width) << 1) | 3];
    if(abs(cr-seedCr)<4){
		int cb=img[(((x + y * width) >> 1) << 2) + 1];
		if(abs(cb-seedCb)<8){
			int cy=img[(x + y * width) << 1];
			atomicAdd(&histY[cy], 1);
		}
    }
}

__global__ void sumGreen(const uint8_t * const img, int w, int h, int width, int height, int pixelSpacing, int seedCr, int seedCb, int seedY, float *data){
    int idx = blockIdx.x*blockDim.x*blockDim.y + (threadIdx.x + threadIdx.y * blockDim.x);
	if( idx >= w*h){
		return;
	}
	int x = (idx - idx/w*w)  * pixelSpacing;
	int y = idx/w * pixelSpacing;
	
	int cy=img[(x + y * width) << 1];
	int cb=img[(((x + y * width) >> 1) << 2) + 1];
	int cr=img[((x + y * width) << 1) | 3];
	atomicAdd(&data[0], cy);
	atomicAdd(&data[2], (cb-128)*(cb-128));
	atomicAdd(&data[3], (cr-128)*(cr-128));
	if(abs(cr-seedCr)<=2&&abs(cb-seedCb)<=2&&abs(cy-seedY)<=2){
		atomicAdd(&data[5], 1);
		if(abs(cr-seedCr)<=1&&abs(cb-seedCb)<=1&&abs(cy-seedY)<=1){
			atomicAdd(&data[4], 1);
		}
	}
}

__global__ void getVarY(const uint8_t * const img, int w, int h, int width, int height, int pixelSpacing, float *data){
    int idx = blockIdx.x*blockDim.x*blockDim.y + (threadIdx.x + threadIdx.y * blockDim.x);
	if( idx >= w*h){
		return;
	}
	int x = (idx - idx/w*w)  * pixelSpacing;
	int y = idx/w * pixelSpacing;
	
	int cy=img[(x + y * width) << 1];
	atomicAdd(&data[1], (cy-data[0])*(cy-data[0]));
}


FieldColorDetector::FieldColorDetector(int _width, int _height, int8_t *_lutCb, int8_t *_lutCr)
    : BaseDetector(_width, _height, _lutCb, _lutCr)
{
	greenCy=0;
	greenCr=0;
	greenCb=0;
	resetArrays();

}

FieldColorDetector::~FieldColorDetector(){
}

/**
 * detects the yCbCr color of the playing field in the image.
 * saves two histograms with rating values for different color combinations
 */
void FieldColorDetector::proceed(const uint8_t * const img) {
	resetArrays();
	searchInitialSeed(img);
	extractFeatures(img,features);
	setYCbCrCube(features);
}
/**
 * dynamic YCbCr-cube size estimation used for green classification
 * (offline training by CMA-ES optimization using 200 labeled color settings)
 */
void FieldColorDetector::setYCbCrCube(float* features){
	int idx=0;
	float minCy=25+50*thetas[idx++];
	float minCb=8+15*thetas[idx++];
	float minCr=8+15*thetas[idx++];
	for(int j=1;j<=NUM_FEATURES;j++){
        float feature=pow(features[j-1],1.3f+thetas[idx++]);
		minCy+=100*thetas[idx++]*feature;
		minCb+=30*thetas[idx++]*feature;
		minCr+=30*thetas[idx++]*feature;
	}
	float gY=3;
	float gC=2;
	if(minCy<1)minCy=1;
	if(minCy>80)minCy=80;
	if(minCb<1)minCb=1;
	if(minCb>30)minCb=30;
	if(minCr<1)minCr=1;
	if(minCr>30)minCr=30;
    float gy=1.5;
    float gc=1.5;
    this->minCy=(int)(greenCy-minCy*gy);
    this->minCb=(int)(greenCb-minCb*gc);
    this->minCr=(int)(greenCr-minCr*gc);
	this->minCy2=(int)(greenCy-minCy*greenGain);
	this->minCb2=(int)(greenCb-minCb*greenGain);
	this->minCr2=(int)(greenCr-minCr*greenGain);

	float maxCy=25+50*thetas[idx++];
	float maxCb=8+15*thetas[idx++];
	float maxCr=8+15*thetas[idx++];
	for(int j=1;j<=NUM_FEATURES;j++){
        float feature=pow(features[j-1],1.3f+thetas[idx++]);
		maxCy+=100*thetas[idx++]*feature;
		maxCb+=30*thetas[idx++]*feature;
		maxCr+=30*thetas[idx++]*feature;
	}
	if(maxCy<1)maxCy=1;
	if(maxCy>80)maxCy=80;
	if(maxCb<1)maxCb=1;
	if(maxCb>30)maxCb=30;
	if(maxCr<1)maxCr=1;
	if(maxCr>30)maxCr=30;
    this->maxCy=(int)(greenCy+maxCy*gy);
    this->maxCb=(int)(greenCb+maxCb*gc);
    this->maxCr=(int)(greenCr+maxCr*gc);
	this->maxCy2=(int)(greenCy+maxCy*greenGain);
	this->maxCb2=(int)(greenCb+maxCb*greenGain);
	this->maxCr2=(int)(greenCr+maxCr*greenGain);
}
/**
 * extraction of image features
 */
void FieldColorDetector::extractFeatures(const uint8_t * const img_d, float* features){

    //uint8_t *img_d;
    //hipMalloc( (void**)&img_d, sizeof(uint8_t) * width * height * 2 );
    //hipMemcpy( img_d, img, sizeof(uint8_t) * width * height * 2, hipMemcpyHostToDevice ) ;
	int cnt=0;

	float meanY=0;
	float varY=0;
	float varCb=0;
	float varCr=0;
	float sumGreen1=0;
	float sumGreen2=0;

    int xi = (width-pixelSpacing / 2) / pixelSpacing;
    int yi = (height-pixelSpacing / 2) / pixelSpacing;
	
	int max = xi*yi;
	int block_num = max / 256 + 1;
	dim3 dim_(16, 16);
	//printf("%d %d max %d, num: %d\n", xi, yi, max, block_num);

	float *data_d, *data = new float[6]; 
	hipMalloc( (void**)&data_d, sizeof(float) * 6 );

	sumGreen<<<block_num, dim_>>>(img_d, xi, yi, width, height, pixelSpacing, seedCr, seedCb, seedY, data_d);
	hipMemcpy( data, data_d, sizeof(float) * 6, hipMemcpyDeviceToHost ) ;
	
	meanY = data[0];
	varCb = data[2];
	varCr = data[3];
	sumGreen1 = data[4];
	sumGreen2 = data[5];
	cnt = max;

	//printf("total %d\n", cnt);
	sumGreen1/=cnt;
	sumGreen2/=cnt;
	varCb=sqrtf(varCb/cnt);
	varCr=sqrtf(varCr/cnt);
	meanY/=cnt;
	data[0] = meanY;

	hipMemcpy( data_d, data, sizeof(float) * 6, hipMemcpyHostToDevice ) ;
	getVarY<<<block_num, dim_>>>(img_d, xi, yi, width, height, pixelSpacing, data_d);
	hipMemcpy( data, data_d, sizeof(float) * 6, hipMemcpyDeviceToHost ) ;
	//printf("total %d, var Y %f %f \n", total, varY, data[1]);
	varY = data[1];
	varY=sqrtf(varY/cnt);
	features[0]=greenCy/256;
	features[1]=varY/32;
	features[2]=varCb/16;
	features[3]=varCr/16;
	features[4]=sumGreen1*50;
	features[5]=sumGreen2*25;
	features[6]=(sumGreen2-sumGreen1)*50;
	//hipFree(img_d);
	hipFree(data_d);
}


/**
 * dominant color search for green detection
 */
void FieldColorDetector::searchInitialSeed(const uint8_t * const img_d){
    //uint8_t *img_d;
    //hipMalloc( (void**)&img_d, sizeof(uint8_t) * width * height * 2 );
    //hipMemcpy( img_d, img, sizeof(uint8_t) * width * height * 2, hipMemcpyHostToDevice ) ;

	//building histogram of all cr-channel
    int seedSearchBorder=width/16;
    int xi = (width-seedSearchBorder * 2) / pixelSpacing;
    int yi = (height -seedSearchBorder * 2) / pixelSpacing;
	
	int max = xi*yi;
	int block_num = max / 256 + 1;
	//printf("%d %d max %d, num: %d\n", xi, yi, max, block_num);

	dim3 dim_(16, 16);

	int *histCr_d;
	hipMalloc( (void**)&histCr_d, 256*sizeof(int) );
    makeHistCr<<<block_num,dim_>>>(img_d, xi, yi, width, height, histCr_d, seedSearchBorder, pixelSpacing);
	hipError_t  error_check ;
	error_check = hipGetLastError();
	if( error_check != hipSuccess ){
	    printf("%s\n" , hipGetErrorString( error_check ) );
	}
	hipMemcpy( histCr, histCr_d, 256*sizeof(int),hipMemcpyDeviceToHost ) ;
	hipFree(histCr_d);
	
    //finding initial cr-value (later used as a seed color)
	seedCr=clamp(colorBorder,getStableMin(histCr,minFieldArea),255-colorBorder);

	//build histogram of cb-channel for promising pixels
    xi = width / pixelSpacing;
	yi = (height) / pixelSpacing;
	
	max = xi*yi;
	block_num = max / 256 + 1;

    int *histCb_d;
    hipMalloc( (void**)&histCb_d, 256*sizeof(int) );
    makeHistCb<<<block_num,dim_>>>(img_d, xi, yi, width, height, histCb_d, seedCr, pixelSpacing);
    hipMemcpy( histCb, histCb_d, 256*sizeof(int),hipMemcpyDeviceToHost ) ;
	hipFree(histCb_d);

	//finding initial cb-value (later used as a seed color)
	seedCb=clamp(colorBorder,getPeak(histCb),255-colorBorder);

	//build histogram of y-channel for promising pixels
    xi = width / pixelSpacing;
	yi = height / pixelSpacing;
	
	max = xi*yi;
	block_num = max / 256 + 1;

    int *histY_d;
    hipMalloc( (void**)&histY_d, 256*sizeof(int) );
    makeHistY<<<block_num,dim_>>>(img_d, xi, yi, width, height, histY_d, seedCr, seedCb, pixelSpacing);
    hipMemcpy( histY, histY_d, 256*sizeof(int),hipMemcpyDeviceToHost ) ;
	hipFree(histY_d);

	//finding initial y-value (later used as a seed color)
	seedY=clamp(colorBorder,getPeak(histY),255-colorBorder);
	greenCy=seedY;
	greenCb=seedCb;
	greenCr=seedCr;
	//hipFree(img_d);
}



void FieldColorDetector::resetArrays() {
    memset(histY,0,sizeof(histY));
    memset(histCb,0,sizeof(histCb));
    memset(histCr,0,sizeof(histCr));
}

/**
 * only in a histogram with 256 bins: get the index of the bin with the lowest value but with a stabilization criteria ('thres').
 */

int FieldColorDetector::getStableMin(const int* const hist, int thres) {
	int sum=0;
	for(int i=0;i<256;i++){
		sum+=hist[i];
		if(sum>thres){
			return i;
		}
	}
	return 0;
}

/**
 * only in a histogram with 256 bins: get the index of the bin with the highest value
 */
int FieldColorDetector::getPeak(const int* const hist) {
	int max=0;
	int maxIdx=0;
	for(int i=0;i<256;i++){
		if(hist[i]>max){
			max=hist[i];
			maxIdx=i;
		}
	}
	return maxIdx;
}

}  // namespace htwk
