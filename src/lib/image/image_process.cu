#include <hip/hip_runtime.h>

__global__ void yuyv2yuv_kernal(unsigned char *in, unsigned char *out, unsigned int w)
{
    int y=blockIdx.x;
    int x=threadIdx.x;
    int tmp = y*w;
    int src_offset = x*2;
    int dst_offset = x*3;
    extern __shared__ unsigned char tp[];
    tp[src_offset+0] = in[tmp*2+src_offset+0];
    tp[src_offset+1] = in[tmp*2+src_offset+1];
    __syncthreads();

    out[tmp*3+dst_offset+0] = tp[src_offset+0];
    out[tmp*3+dst_offset+1] = tp[src_offset+(int)pow(-1, x&1)];
    out[tmp*3+dst_offset+2] = tp[src_offset+2+(int)pow(-1, x&1)];
}

template <typename T>
__device__ T boundrgb(T v)
{
    if(v>255) return 255;
    if(v<0) return 0;
    return v;
}

__global__ void yuyv2bgr_kernal(unsigned char *in, unsigned char *out, unsigned int w)
{
    int y=blockIdx.x;
    int x=threadIdx.x;
    int tmp = y*w;
    int src_offset = x*2;
    int dst_offset = x*3;
    extern __shared__ unsigned char tp[];
    tp[src_offset+0] = in[tmp*2+src_offset+0];
    tp[src_offset+1] = in[tmp*2+src_offset+1];
    __syncthreads();

    unsigned char Y = tp[src_offset+0];
    unsigned char U = tp[src_offset+(int)pow(-1, x&1)];
    unsigned char V = tp[src_offset+2+(int)pow(-1, x&1)];
    int r,g,b;

    r = (1.164 * (Y - 16)) + (2.018 * (V - 128));
    g = (1.164 * (Y - 16)) - (0.813 * (U - 128)) - (0.391 * (V - 128));
    b = (1.164 * (Y - 16)) + (1.596 * (U - 128));
    r = boundrgb<int>(r);
    g = boundrgb<int>(g);
    b = boundrgb<int>(b);

    out[tmp*3+dst_offset+2] = (unsigned char)(r);
    out[tmp*3+dst_offset+1] = (unsigned char)(g);
    out[tmp*3+dst_offset+0] = (unsigned char)(b);
}

__global__ void yuyv2rgbpf_kernal(unsigned char *in, float *out, unsigned int w, unsigned int h)
{
    int y=blockIdx.x;
    int x=threadIdx.x;
    int planesize = w*h;
    int tmp = y*w;
    int src_offset = x*2;
    extern __shared__ unsigned char tp[];
    tp[src_offset+0] = in[tmp*2+src_offset+0];
    tp[src_offset+1] = in[tmp*2+src_offset+1];
    __syncthreads();

    unsigned char Y = tp[src_offset+0];
    unsigned char U = tp[src_offset+(int)pow(-1, x&1)];
    unsigned char V = tp[src_offset+2+(int)pow(-1, x&1)];
    float r,g,b;

    r = (1.164 * (Y - 16)) + (2.018 * (V - 128));
    g = (1.164 * (Y - 16)) - (0.813 * (U - 128)) - (0.391 * (V - 128));
    b = (1.164 * (Y - 16)) + (1.596 * (U - 128));
    r = boundrgb<float>(r)/255.0f;
    g = boundrgb<float>(g)/255.0f;
    b = boundrgb<float>(b)/255.0f;

    out[tmp+x] = r;
    out[planesize+tmp+x] = g;
    out[2*planesize+tmp+x] = b;
}

__global__ void yuyv2all_kernal(unsigned char *in, unsigned char *yuv, unsigned char *bgr,
        float *rgbpf, unsigned int w, unsigned int h)
{
    int y=blockIdx.x;
    int x=threadIdx.x;
    int planesize = w*h;
    int tmp = y*w;
    int src_offset = x*2;
    int dst_offset = x*3;
    extern __shared__ unsigned char tp[];
    tp[src_offset+0] = in[tmp*2+src_offset+0];
    tp[src_offset+1] = in[tmp*2+src_offset+1];
    __syncthreads();

    unsigned char Y = tp[src_offset+0];
    unsigned char U = tp[src_offset+(int)pow(-1, x&1)];
    unsigned char V = tp[src_offset+2+(int)pow(-1, x&1)];
    yuv[tmp*3+dst_offset+0] = Y;
    yuv[tmp*3+dst_offset+1] = U;
    yuv[tmp*3+dst_offset+2] = V;

    float r,g,b;
    r = (1.164 * (Y - 16)) + (2.018 * (V - 128));
    g = (1.164 * (Y - 16)) - (0.813 * (U - 128)) - (0.391 * (V - 128));
    b = (1.164 * (Y - 16)) + (1.596 * (U - 128));
    r = boundrgb<float>(r);
    g = boundrgb<float>(g);
    b = boundrgb<float>(b);
    bgr[tmp*3+dst_offset+0] = (unsigned char)b;
    bgr[tmp*3+dst_offset+1] = (unsigned char)g;
    bgr[tmp*3+dst_offset+2] = (unsigned char)r;

    rgbpf[tmp+x] = r/255.0f;
    rgbpf[planesize+tmp+x] = g/255.0f;
    rgbpf[2*planesize+tmp+x] = b/255.0f;
}

__global__ void yuyv2dst_kernal(unsigned char *in, unsigned char *bgr, float *rgb, unsigned int w, unsigned int h)
{
    int y=blockIdx.x;
    int x=threadIdx.x;
    int planesize = w*h;
    int tmp = y*w;
    int src_offset = x*2;
    int dst_offset = x*3;
    extern __shared__ unsigned char tp[];
    tp[src_offset+0] = in[tmp*2+src_offset+0];
    tp[src_offset+1] = in[tmp*2+src_offset+1];
    __syncthreads();

    unsigned char Y = tp[src_offset+0];
    unsigned char U = tp[src_offset+(int)pow(-1, x&1)];
    unsigned char V = tp[src_offset+2+(int)pow(-1, x&1)];


    float r,g,b;
    r = (1.164 * (Y - 16)) + (2.018 * (V - 128));
    g = (1.164 * (Y - 16)) - (0.813 * (U - 128)) - (0.391 * (V - 128));
    b = (1.164 * (Y - 16)) + (1.596 * (U - 128));
    r = boundrgb<float>(r);
    g = boundrgb<float>(g);
    b = boundrgb<float>(b);
    bgr[tmp*3+dst_offset+0] = (unsigned char)b;
    bgr[tmp*3+dst_offset+1] = (unsigned char)g;
    bgr[tmp*3+dst_offset+2] = (unsigned char)r;

    rgb[tmp+x] = r/255.0f;
    rgb[planesize+tmp+x] = g/255.0f;
    rgb[2*planesize+tmp+x] = b/255.0f;
}

__global__ void resizew_kernal(float *src, int iw, int ih, float *dst, int ow, int oh)
{
    int y = blockIdx.x;
    int x = threadIdx.x;
    int planesizeo = ow*oh;
    int planesizei = iw*ih;
    int tmpo = y*ow;
    int tmpi = y*iw;
    float w_scale = (float)(iw-1)/(ow-1);
    float sx = x*w_scale;
    int ix = (int) sx;
    float dx = sx - ix;
    if(ix<iw-1)
    {
        dst[tmpo+x] = (1-dx)*src[tmpi+ix]+dx*src[tmpi+ix+1];
        dst[planesizeo+tmpo+x] = (1-dx)*src[planesizei+tmpi+ix]+dx*src[planesizei+tmpi+ix+1];
        dst[2*planesizeo+tmpo+x] = (1-dx)*src[2*planesizei+tmpi+ix]+dx*src[2*planesizei+tmpi+ix+1];
    }
}

__global__ void resizeh_kernal(float *src, int iw, int ih, float *dst, int ow, int oh)
{
    int y = blockIdx.x;
    int x = threadIdx.x;
    int planesizeo = ow*oh;
    int planesizei = iw*ih;
    int tmpo = y*ow;
    float h_scale = (float)(ih-1)/(oh-1);

    float sy = y*h_scale;
    int iy = (int) sy;
    float dy = sy - iy;
    int tmpi1 = iy*iw;
    int tmpi2 = (iy+1)*iw;
    if(iy<ih-1)
    {
        dst[tmpo+x] = (1-dy)*src[tmpi1+x]+dy*src[tmpi2+x];
        dst[planesizeo+tmpo+x] = (1-dy)*src[planesizei+tmpi1+x]+dy*src[planesizei+tmpi2+x];
        dst[2*planesizeo+tmpo+x] = (1-dy)*src[2*planesizei+tmpi1+x]+dy*src[2*planesizei+tmpi2+x];
    }
}

void cudaYUYV2YUV(unsigned char *in, unsigned char *out, const unsigned int &w, const unsigned int &h)
{
    yuyv2yuv_kernal<<<h, w, w*2>>>(in,out,w);
}

void cudaYUYV2BGR(unsigned char *in, unsigned char *out, const unsigned int &w, const unsigned int &h)
{
    yuyv2bgr_kernal<<<h, w, w*2>>>(in,out,w);
}

void cudaYUYV2RGBPF(unsigned char *in, float *out, const unsigned int &w, const unsigned int &h)
{
    yuyv2rgbpf_kernal<<<h, w, w*2>>>(in,out,w,h);
}

void cudaYUYV2DST(unsigned char *in, unsigned char *bgr, float *rgb, const unsigned int &w, const unsigned int &h)
{
    yuyv2dst_kernal<<<h,w,w*2>>>(in, bgr, rgb, w, h);
}

void cudaResize(float *in, int iw, int ih, float *sizedw, float *sized, int ow, int oh)
{
    resizew_kernal<<<ih, ow>>>(in, iw, ih, sizedw, ow, ih);
    resizeh_kernal<<<oh, ow>>>(sizedw, ow, ih, sized, ow, oh);
}